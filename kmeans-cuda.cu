#include "hip/hip_runtime.h"
#include "kmeans.h"

#include <hip/hip_runtime_api.h>
#ifdef PROFILE
#include <hip/hip_runtime_api.h>
#endif

__global__ static 
void find_nearest_cluster(
    double **points,
    double **centroids,
    int num_points,
    int num_coords,
    int num_centroids) {
  extern __shared__ char shared[]; // array of bytes of shared memory

  int point_id = blockDim.x * blockIdx.x + threadIdx.x;
  printf("%d\n", point_id);
}

void kmeans(
    double **points,
    double **centroids,
    double **old_centroids,
    int num_points,
    int num_coords,
    int num_centroids,
    int max_iterations,
    double threshold,
    int workers) {

  double *dev_points;
  double *dev_centroids;

  const size_t threads_per_block = 128; // This is a design decision
  const size_t num_blocks = (num_points + threads_per_block - 1) / threads_per_block;
  const size_t shared_mem_per_block = threads_per_block * sizeof(char);

  cout << "kmeans" << endl;

  hipMalloc(&dev_points, num_points * num_coords, * sizeof(double));
  hipMalloc(&dev_centroids, num_centroids * num_coords, * sizeof(double));

  find_nearest_cluster
    <<<num_blocks, threads_per_block, shared_mem_per_block>>>
    (points, centroids, num_points, num_coords, num_centroids);

  hipDeviceSynchronize();
}
